#include "hip/hip_runtime.h"
#include "cuda_model_tf.h"

/*------------------------------------------------------
* Kernel Forward Declarations 
*------------------------------------------------------*/
__global__ void square(float *x, float *y);

/*------------------------------------------------------
* STUB CALLS
*------------------------------------------------------*/
void TF_CPU_testSquare()
{
	const int N = 10000;
	float x[N], y[N];


	//Populate with some dummy data
	for (int i = 0; i < N; i++)
	{
		x[i] = 3.34;
		y[i] = 0.0;
	}
		

	//Figure out the size and then square away!
	for (int i = 0; i < N; i++)
		y[i] = x[i] * x[i];
}

void TF_GPU_testSquare(int blocks, int threads, float *d_x, float *d_y)
{
	square<<<blocks, threads>>>(d_x, d_y);
}


/*------------------------------------------------------
* GLOBAL KERNELS:
* Caller	-->	CPU
* Execute	-->	GPU
*------------------------------------------------------*/
__global__ void square(float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	y[i] = x[i] * x[i];
}

/*------------------------------------------------------
* LOCAL KERNELS:
* Caller	-->	GPU
* Execute	-->	GPU
*------------------------------------------------------*/