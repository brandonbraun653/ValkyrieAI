#include "cuda_ga.h"


//Eventually I'll add some more things here.
void GPU_DisplayHeader()
{
	const int kb = 1024;
	const int mb = kb * kb;
	std::cout << "NBody.GPU" << std::endl << "=========" << std::endl << std::endl;

	std::wcout << "CUDA Version:   v" << CUDART_VERSION << std::endl;
	//std::wcout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << std::endl << std::endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	std::wcout << "CUDA Devices: " << devCount << std::endl << std::endl;

	for (int i = 0; i < devCount; ++i)
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		std::wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
		std::wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << std::endl;
		std::wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << std::endl;
		std::wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << std::endl;
		std::wcout << "  Block registers: " << props.regsPerBlock << std::endl << std::endl;

		std::wcout << "  Warp size:         " << props.warpSize << std::endl;
		std::wcout << "  Threads per block: " << props.maxThreadsPerBlock << std::endl;
		std::wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
		std::wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << std::endl;
		std::wcout << std::endl;
	}
}
